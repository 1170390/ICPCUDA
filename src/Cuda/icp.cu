#include "hip/hip_runtime.h"
#include "internal.h"
#include "containers/safe_call.hpp"

#if __CUDA_ARCH__ < 300
__inline__ __device__
float __shfl_down(float val, int offset, int width = 32)
{
    static __shared__ float shared[MAX_THREADS];
    int lane = threadIdx.x % 32;
    shared[threadIdx.x] = val;
    __syncthreads();
    val = (lane + offset < width) ? shared[threadIdx.x + offset] : 0;
    __syncthreads();
    return val;
}
#endif

#if __CUDA_ARCH__ < 350
template<typename T>
__device__ __forceinline__ T __ldg(const T* ptr)
{
    return *ptr;
}
#endif

__inline__  __device__ jtjjtr warpReduceSum(jtjjtr val)
{
    for(int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        val.aa += __shfl_down(val.aa, offset);
        val.ab += __shfl_down(val.ab, offset);
        val.ac += __shfl_down(val.ac, offset);
        val.ad += __shfl_down(val.ad, offset);
        val.ae += __shfl_down(val.ae, offset);
        val.af += __shfl_down(val.af, offset);
        val.ag += __shfl_down(val.ag, offset);

        val.bb += __shfl_down(val.bb, offset);
        val.bc += __shfl_down(val.bc, offset);
        val.bd += __shfl_down(val.bd, offset);
        val.be += __shfl_down(val.be, offset);
        val.bf += __shfl_down(val.bf, offset);
        val.bg += __shfl_down(val.bg, offset);

        val.cc += __shfl_down(val.cc, offset);
        val.cd += __shfl_down(val.cd, offset);
        val.ce += __shfl_down(val.ce, offset);
        val.cf += __shfl_down(val.cf, offset);
        val.cg += __shfl_down(val.cg, offset);

        val.dd += __shfl_down(val.dd, offset);
        val.de += __shfl_down(val.de, offset);
        val.df += __shfl_down(val.df, offset);
        val.dg += __shfl_down(val.dg, offset);

        val.ee += __shfl_down(val.ee, offset);
        val.ef += __shfl_down(val.ef, offset);
        val.eg += __shfl_down(val.eg, offset);

        val.ff += __shfl_down(val.ff, offset);
        val.fg += __shfl_down(val.fg, offset);

        val.residual += __shfl_down(val.residual, offset);
        val.inliers += __shfl_down(val.inliers, offset);
    }

    return val;
}

__inline__  __device__ jtjjtr blockReduceSum(jtjjtr val)
{
    static __shared__ jtjjtr shared[32];

    int lane = threadIdx.x % warpSize;

    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);

    //write reduced value to shared memory
    if(lane == 0)
    {
        shared[wid] = val;
    }
    __syncthreads();

    const jtjjtr zero = {0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0, 0, 0, 0,
                         0, 0, 0, 0, 0};

    //ensure we only grab a value from shared memory if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : zero;

    if(wid == 0)
    {
        val = warpReduceSum(val);
    }

    return val;
}

__global__ void reduceSum(jtjjtr * in, jtjjtr * out, int N)
{
    jtjjtr sum = {0, 0, 0, 0, 0, 0, 0, 0,
                  0, 0, 0, 0, 0, 0, 0, 0,
                  0, 0, 0, 0, 0, 0, 0, 0,
                  0, 0, 0, 0, 0};

    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        sum.add(in[i]);
    }

    sum = blockReduceSum(sum);

    if(threadIdx.x == 0)
    {
        out[blockIdx.x] = sum;
    }
}

struct ICPReduction
{
    Eigen::Matrix<float, 3, 3, Eigen::DontAlign> R_prev_curr;
    Eigen::Matrix<float, 3, 1, Eigen::DontAlign> t_prev_curr;

    PtrStep<float> vmap_curr;
    PtrStep<float> nmap_curr;

    Intr intr;

    PtrStep<float> vmap_prev;
    PtrStep<float> nmap_prev;

    float distThres;
    float angleThres;

    int cols;
    int rows;
    int N;

    jtjjtr * out;

    __device__ __forceinline__ bool
    search (const int & x,
            const int & y,
            Eigen::Matrix<float,3,1,Eigen::DontAlign>& n,
            Eigen::Matrix<float,3,1,Eigen::DontAlign>& d,
            Eigen::Matrix<float,3,1,Eigen::DontAlign>& s) const
    {
        const Eigen::Matrix<float,3,1,Eigen::DontAlign> v_curr(vmap_curr.ptr(y)[x],
                                                               vmap_curr.ptr(y + rows)[x],
                                                               vmap_curr.ptr(y + 2 * rows)[x]);

        const Eigen::Matrix<float,3,1,Eigen::DontAlign> v_curr_in_prev = R_prev_curr * v_curr + t_prev_curr;

        const Eigen::Matrix<int,2,1,Eigen::DontAlign> p_curr_in_prev(__float2int_rn(v_curr_in_prev(0) * intr.fx / v_curr_in_prev(2) + intr.cx),
                                                                     __float2int_rn(v_curr_in_prev(1) * intr.fy / v_curr_in_prev(2) + intr.cy));

        if(p_curr_in_prev(0) < 0 || p_curr_in_prev(1) < 0 || p_curr_in_prev(0) >= cols || p_curr_in_prev(1) >= rows || v_curr_in_prev(2) < 0)
            return false;

        const Eigen::Matrix<float,3,1,Eigen::DontAlign> v_prev(__ldg(&vmap_prev.ptr(p_curr_in_prev(1))[p_curr_in_prev(0)]),
                                                               __ldg(&vmap_prev.ptr(p_curr_in_prev(1) + rows)[p_curr_in_prev(0)]),
                                                               __ldg(&vmap_prev.ptr(p_curr_in_prev(1) + 2 * rows)[p_curr_in_prev(0)]));

        const Eigen::Matrix<float,3,1,Eigen::DontAlign> n_curr(nmap_curr.ptr(y)[x],
                                                               nmap_curr.ptr(y + rows)[x],
                                                               nmap_curr.ptr(y + 2 * rows)[x]);

        const Eigen::Matrix<float,3,1,Eigen::DontAlign> n_curr_in_prev = R_prev_curr * n_curr;

        const Eigen::Matrix<float,3,1,Eigen::DontAlign> n_prev(__ldg(&nmap_prev.ptr(p_curr_in_prev(1))[p_curr_in_prev(0)]),
                                                               __ldg(&nmap_prev.ptr(p_curr_in_prev(1) + rows)[p_curr_in_prev(0)]),
                                                               __ldg(&nmap_prev.ptr(p_curr_in_prev(1) + 2 * rows)[p_curr_in_prev(0)]));

        const float dist = (v_prev - v_curr_in_prev).norm();
        const float sine = n_curr_in_prev.cross(n_prev).norm();

        n = n_prev;
        d = v_prev;
        s = v_curr_in_prev;

        return (sine < angleThres && dist <= distThres && !isnan(n_curr(0)) && !isnan(n_prev(0)));
    }

    __device__ __forceinline__ jtjjtr
    getProducts(int & i) const
    {
        int y = i / cols;
        int x = i - (y * cols);

        Eigen::Matrix<float,3,1,Eigen::DontAlign> n, d, s;

        bool found_coresp = search(x, y, n, d, s);

        float row[7] = {0, 0, 0, 0, 0, 0, 0};

        if(found_coresp)
        {
            *(Eigen::Matrix<float,3,1,Eigen::DontAlign>*)&row[0] = n;
            *(Eigen::Matrix<float,3,1,Eigen::DontAlign>*)&row[3] = s.cross(n);
            row[6] = n.dot(d - s);
        }

        jtjjtr values = {row[0] * row[0],
                         row[0] * row[1],
                         row[0] * row[2],
                         row[0] * row[3],
                         row[0] * row[4],
                         row[0] * row[5],
                         row[0] * row[6],

                         row[1] * row[1],
                         row[1] * row[2],
                         row[1] * row[3],
                         row[1] * row[4],
                         row[1] * row[5],
                         row[1] * row[6],

                         row[2] * row[2],
                         row[2] * row[3],
                         row[2] * row[4],
                         row[2] * row[5],
                         row[2] * row[6],

                         row[3] * row[3],
                         row[3] * row[4],
                         row[3] * row[5],
                         row[3] * row[6],

                         row[4] * row[4],
                         row[4] * row[5],
                         row[4] * row[6],

                         row[5] * row[5],
                         row[5] * row[6],

                         row[6] * row[6],
                         found_coresp};

        return values;
    }

    __device__ __forceinline__ void
    operator () () const
    {
        jtjjtr sum = {0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0, 0, 0, 0,
                      0, 0, 0, 0, 0};

        for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        {
            jtjjtr val = getProducts(i);

            sum.add(val);
        }

        sum = blockReduceSum(sum);

        if(threadIdx.x == 0)
        {
            out[blockIdx.x] = sum;
        }
    }
};

__global__ void icpKernel(const ICPReduction icp)
{
    icp();
}

void icpStep(const Eigen::Matrix<float,3,3,Eigen::DontAlign> & R_prev_curr,
             const Eigen::Matrix<float,3,1,Eigen::DontAlign> & t_prev_curr,
             const DeviceArray2D<float>& vmap_curr,
             const DeviceArray2D<float>& nmap_curr,
             const Intr& intr,
             const DeviceArray2D<float>& vmap_prev,
             const DeviceArray2D<float>& nmap_prev,
             float distThres,
             float angleThres,
             DeviceArray<jtjjtr> & sum,
             DeviceArray<jtjjtr> & out,
             float * matrixA_host,
             float * vectorB_host,
             float * residual_host,
             int threads, int blocks)
{
    int cols = vmap_curr.cols ();
    int rows = vmap_curr.rows () / 3;

    ICPReduction icp;

    icp.R_prev_curr = R_prev_curr;
    icp.t_prev_curr = t_prev_curr;

    icp.vmap_curr = vmap_curr;
    icp.nmap_curr = nmap_curr;

    icp.intr = intr;

    icp.vmap_prev = vmap_prev;
    icp.nmap_prev = nmap_prev;

    icp.distThres = distThres;
    icp.angleThres = angleThres;

    icp.cols = cols;
    icp.rows = rows;

    icp.N = cols * rows;
    icp.out = sum;

    icpKernel<<<blocks, threads>>>(icp);

    reduceSum<<<1, MAX_THREADS>>>(sum, out, blocks);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());

    float host_data[32];
    out.download((jtjjtr *)&host_data[0]);

    int shift = 0;
    for (int i = 0; i < 6; ++i)  //rows
    {
        for (int j = i; j < 7; ++j)    // cols + b
        {
            float value = host_data[shift++];
            if (j == 6)       // vector b
                vectorB_host[i] = value;
            else
                matrixA_host[j * 6 + i] = matrixA_host[i * 6 + j] = value;
        }
    }

    residual_host[0] = host_data[27];
    residual_host[1] = host_data[28];
}
